
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel-execution with __global__: empty function at this point

__global__ void kernel(void) {
// printf("Hello, Cuda!\n");
}

int main(void) {
// Kernel execution with <<<1,1>>>
kernel<<<1,1>>>();
printf("Hello, World!\n");
return 0;
}
